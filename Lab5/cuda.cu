#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <ctime>
typedef unsigned char uchar;

#define EROSION_SIZE 5
const int EROSION_MATRIX[EROSION_SIZE * EROSION_SIZE] = {
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0,
    1, 1, 1, 1, 1,
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0
};
#define EMPTY 0
#define FULL 255
#define THREAD_CNT 1024

__global__ void erosion_kernel(uchar *image, uchar *res, int *matrix, int rows, int cols) {
    int i = threadIdx.x;
    int begin = i * rows / THREAD_CNT;
    int end = min((i + 1) * rows / THREAD_CNT, rows);
    for (int pos = begin; pos < end; ++pos) {
        uchar* aim = res + (pos * cols);
        if (pos < EROSION_SIZE / 2 || pos >= rows - EROSION_SIZE / 2) {
            for (int l = 0; l < cols; ++l)
                aim[l] = FULL;
            continue;
        }
        uchar* cache[EROSION_SIZE];
        for (int i = 0; i < EROSION_SIZE; ++i) {
            cache[i] = image + (i + pos - EROSION_SIZE / 2) * cols;
        }
        int l;
        for (l = 0; l < EROSION_SIZE / 2 && l < cols; ++l) {
            aim[l] = FULL;
        }
        for (; l < cols - EROSION_SIZE / 2; ++l) {
            bool result = true;
            for (int i = 0; i < EROSION_SIZE && result; ++i) {
                for (int j = 0; j < EROSION_SIZE; ++j) {
                    if (matrix[i * EROSION_SIZE + j] && !cache[i][j + l - EROSION_SIZE / 2]) {
                        result = false;
                        break;
                    }
                }
            }
            if (result) aim[l] = FULL;
            else aim[l] = EMPTY;
        }
        for (; l < cols; ++l)
            aim[l] = FULL;
    }
}

void erosion_image(uchar *image_data, uchar *res_data, int rows, int cols) {
    int size = rows * cols;

    int *dev_matrix = 0;
    uchar *dev_image_data = 0;
    uchar *dev_res_data = 0;
    clock_t begin;
    clock_t end;
    double timeCost;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_image_data, size * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_res_data, size * sizeof(uchar));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_matrix, EROSION_SIZE * EROSION_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_image_data, image_data, size * sizeof(uchar), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // cudaStatus = cudaMemcpy(dev_res_data, res_data, size * sizeof(uchar), cudaMemcpyHostToDevice);
    // if (cudaStatus != cudaSuccess) {
    //     fprintf(stderr, "cudaMalloc failed!");
    //     goto Error;
    // }

    cudaStatus = hipMemcpy(dev_matrix, EROSION_MATRIX, EROSION_SIZE * EROSION_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    begin = clock();
    erosion_kernel<<<1, THREAD_CNT>>>(dev_image_data, dev_res_data, dev_matrix, rows, cols);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    end = clock();
    timeCost = (double)(end - begin)/CLOCKS_PER_SEC;
    fprintf(stderr, "time: %lf\n", timeCost);

    cudaStatus = hipMemcpy(res_data, dev_res_data, size * sizeof(uchar), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

Error:
    hipFree(dev_image_data);
    hipFree(dev_res_data);
    hipFree(dev_matrix);

    hipDeviceReset();
}