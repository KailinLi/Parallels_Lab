#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

typedef unsigned char uchar;

#define EROSION_SIZE 5
const int EROSION_MATRIX[EROSION_SIZE * EROSION_SIZE] = {
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0,
    1, 1, 1, 1, 1,
    0, 0, 1, 0, 0,
    0, 0, 1, 0, 0
};
#define EMPTY 0
#define FULL 255
#define THREAD_CNT 20


__global__ void erosion_line(uchar* image, uchar* res, int *matrix, int pos, int rows, int cols) {
    uchar* aim = res + (pos * cols);
    if (pos < EROSION_SIZE / 2 || pos >= rows - EROSION_SIZE / 2) {
        for (int l = 0; l < cols; ++l)
            aim[l] = FULL;
        return;
    }
    uchar* cache[EROSION_SIZE];
    for (int i = 0; i < EROSION_SIZE; ++i) {
        cache[i] = image + (i + pos - EROSION_SIZE / 2) * cols;
    }
    int l;
    for (l = 0; l < EROSION_SIZE / 2 && l < cols; ++l) {
        aim[l] = FULL;
    }
    for (; l < cols - EROSION_SIZE / 2; ++l) {
        bool result = true;
        for (int i = 0; i < EROSION_SIZE && result; ++i) {
            for (int j = 0; j < EROSION_SIZE; ++j) {
                if (matrix[i * EROSION_SIZE + j] && !cache[i][j + l - EROSION_SIZE / 2]) {
                    result = false;
                    break;
                }
            }
        }
        if (result) aim[l] = FULL;
        else aim[l] = EMPTY;
    }
    for (; l < cols; ++l)
        aim[l] = FULL;
}

__global__ void erosion_kernel(uchar *image, uchar *res, int *matrix, int rows, int cols) {
    int i = threadIdx.x;
    int begin = i * rows / THREAD_CNT;
    int end = min((i + 1) * rows / THREAD_CNT, rows);
    for (int pos = begin; pos < end; ++pos) {
        erosion_line(image, res, matrix, k, rows, cols);
    }
}

void erosion_image(uchar *image_data, uchar *res_data, int rows, int cols) {
    int size = rows * cols;

    int *dev_matrix = 0;
    uchar *dev_image_data = 0;
    uchar *dev_res_data = 0;

    hipSetDevice(0);
    hipMalloc((void**)&dev_image_data, size * sizeof(uchar));
    hipMalloc((void**)&dev_res_data, size * sizeof(uchar));
    hipMalloc((void**)&dev_matrix, EROSION_SIZE * EROSION_SIZE * sizeof(int));

    hipMemcpy(dev_image_data, image_data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(dev_res_data, res_data, size * sizeof(uchar), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrix, EROSION_MATRIX, EROSION_SIZE * EROSION_SIZE * sizeof(uchar), hipMemcpyHostToDevice);

    erosion_kernel<<<1, THREAD_CNT>>>(dev_image_data, dev_res_data, dev_matrix, rows, cols);

    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(res_data, dev_res_data, size * sizeof(uchar), hipMemcpyDeviceToHost);

    hipFree(dev_image_data);
    hipFree(dev_res_data);
    hipFree(dev_matrix);
}