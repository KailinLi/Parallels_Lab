#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
const int SIZE = 5;

void addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
void init_vector(int *v) {
    for (int i = 0; i < N; ++i)
        v[i] = i;
}
int main() {
    const int a[SIZE];
    const int b[SIZE];
    int c[SIZE];
    init_vector(a);
    init_vector(b);
    addWithCuda(c, a, b, SIZE);
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", a[i]);
    printf("} + ");
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", b[i]);
    printf("} =\n");
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", c[i]);
    printf("}\n");

    hipDeviceReset();

    return 0;
}

void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipSetDevice(0);

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return;
}
