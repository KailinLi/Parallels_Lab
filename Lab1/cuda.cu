#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
const int SIZE = 5;

void addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
void init_vector(int *v) {
    for (int i = 0; i < N; ++i)
        v[i] = i;
}
int main() {
    const int a[SIZE];
    const int b[SIZE];
    int c[SIZE];
    init_vector(a);
    init_vector(b);
    addWithCuda(c, a, b, SIZE);
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", a[i]);
    printf("} + ");
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", b[i]);
    printf("} =\n");
    printf("{");
    for (int i = 0; i < SIZE; ++i)
        printf("%d, ", c[i]);
    printf("}\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMalloc((void**)&dev_a, size * sizeof(int));

    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return;
}
